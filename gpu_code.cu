#include "hip/hip_runtime.h"
//anatara Arun  Natarajan
#include <unistd.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdlib.h>
#include <sys/time.h>
#include "rta.h"
#include <sstream>
#include <iostream>




#define SHA1CircularShift(bits,word) \
                ((((word) << (bits)) & 0xFFFFFFFF) | \
                ((word) >> (32-(bits))))


typedef struct sha1calc {
     unsigned    temp;              
     unsigned    W[80];         
     unsigned    A, B, C, D, E;
}sha1calc;

__device__  void SHA1Reset(SHA1Context *context)
{
    context->Length_Low             = 0;
    context->Length_High            = 0;
    context->Message_Block_Index    = 0;

    context->Message_Digest[0]      = 0x67452301;
    context->Message_Digest[1]      = 0xEFCDAB89;
    context->Message_Digest[2]      = 0x98BADCFE;
    context->Message_Digest[3]      = 0x10325476;
    context->Message_Digest[4]      = 0xC3D2E1F0;

    context->Computed   = 0;
    context->Corrupted  = 0;
}

__device__  void SHA1ProcessMessageBlock(SHA1Context *context, int tix)
{
    const unsigned K[] =            /* Constants defined in SHA-1   */      
    {
        0x5A827999,
        0x6ED9EBA1,
        0x8F1BBCDC,
        0xCA62C1D6
    };
     int         t;                  /* Loop counter                 */
	__shared__ sha1calc scalc[RECEIVERS];

    /*
     *  Initialize the first 16 words in the array W
     */
    for(t = 0; t < 16; t++)
    {
        scalc[tix].W[t] = ((unsigned) context->Message_Block[t * 4]) << 24;
        scalc[tix].W[t] |= ((unsigned) context->Message_Block[t * 4 + 1]) << 16;
        scalc[tix].W[t] |= ((unsigned) context->Message_Block[t * 4 + 2]) << 8;
        scalc[tix].W[t] |= ((unsigned) context->Message_Block[t * 4 + 3]);
    }

    for(t = 16; t < 80; t++)
    {
       scalc[tix].W[t] = SHA1CircularShift(1,scalc[tix].W[t-3] ^ scalc[tix].W[t-8] ^ scalc[tix].W[t-14] ^ scalc[tix].W[t-16]);
    }

    scalc[tix].A = context->Message_Digest[0];
    scalc[tix].B = context->Message_Digest[1];
    scalc[tix].C = context->Message_Digest[2];
    scalc[tix].D = context->Message_Digest[3];
    scalc[tix].E = context->Message_Digest[4];

    for(t = 0; t < 20; t++)
    {
        scalc[tix].temp =  SHA1CircularShift(5,scalc[tix].A) +
                ((scalc[tix].B & scalc[tix].C) | ((~scalc[tix].B) & scalc[tix].D)) + scalc[tix].E + scalc[tix].W[t] + K[0];
        scalc[tix].temp &= 0xFFFFFFFF;
        scalc[tix].E = scalc[tix].D;
        scalc[tix].D = scalc[tix].C;
        scalc[tix].C = SHA1CircularShift(30,scalc[tix].B);
        scalc[tix].B = scalc[tix].A;
        scalc[tix].A = scalc[tix].temp;
    }

    for(t = 20; t < 40; t++)
    {
        scalc[tix].temp = SHA1CircularShift(5,scalc[tix].A) + (scalc[tix].B ^ scalc[tix].C ^ scalc[tix].D) + scalc[tix].E + scalc[tix].W[t] + K[1];
        scalc[tix].temp &= 0xFFFFFFFF;
        scalc[tix].E = scalc[tix].D;
        scalc[tix].D = scalc[tix].C;
        scalc[tix].C = SHA1CircularShift(30,scalc[tix].B);
        scalc[tix].B = scalc[tix].A;
        scalc[tix].A = scalc[tix].temp;
    }

    for(t = 40; t < 60; t++)
    {
        scalc[tix].temp = SHA1CircularShift(5,scalc[tix].A) +
               ((scalc[tix].B & scalc[tix].C) | (scalc[tix].B & scalc[tix].D) | (scalc[tix].C & scalc[tix].D)) + scalc[tix].E + scalc[tix].W[t] + K[2];
        scalc[tix].temp &= 0xFFFFFFFF;
        scalc[tix].E = scalc[tix].D;
        scalc[tix].D = scalc[tix].C;
        scalc[tix].C = SHA1CircularShift(30,scalc[tix].B);
        scalc[tix].B = scalc[tix].A;
        scalc[tix].A = scalc[tix].temp;
    }

    for(t = 60; t < 80; t++)
    {
        scalc[tix].temp = SHA1CircularShift(5,scalc[tix].A) + (scalc[tix].B ^ scalc[tix].C ^ scalc[tix].D) + scalc[tix].E + scalc[tix].W[t] + K[3];
        scalc[tix].temp &= 0xFFFFFFFF;
        scalc[tix].E = scalc[tix].D;
        scalc[tix].D = scalc[tix].C;
        scalc[tix].C = SHA1CircularShift(30,scalc[tix].B);
        scalc[tix].B = scalc[tix].A;
        scalc[tix].A = scalc[tix].temp;
    }

    context->Message_Digest[0] =
                        (context->Message_Digest[0] + scalc[tix].A) & 0xFFFFFFFF;
    context->Message_Digest[1] =
                        (context->Message_Digest[1] + scalc[tix].B) & 0xFFFFFFFF;
    context->Message_Digest[2] =
                        (context->Message_Digest[2] + scalc[tix].C) & 0xFFFFFFFF;
    context->Message_Digest[3] =
                        (context->Message_Digest[3] + scalc[tix].D) & 0xFFFFFFFF;
    context->Message_Digest[4] =
                        (context->Message_Digest[4] + scalc[tix].E) & 0xFFFFFFFF;

    context->Message_Block_Index = 0;
}

__device__  void SHA1Input(SHA1Context *context, char *message_array, unsigned length, int tix)
{
    if (!length)
    {
        return;
    }

    if (context->Computed || context->Corrupted)
    {
        context->Corrupted = 1;
        return;
    }

    while(length-- && !context->Corrupted)
    {
        context->Message_Block[context->Message_Block_Index++] =
                                                (*message_array & 0xFF);

        context->Length_Low += 8;
        /* Force it to 32 bits */
        context->Length_Low &= 0xFFFFFFFF;
        if (context->Length_Low == 0)
        {
            context->Length_High++;
            /* Force it to 32 bits */
            context->Length_High &= 0xFFFFFFFF;
            if (context->Length_High == 0)
            {
                /* Message is too long */
                context->Corrupted = 1;
            }
        }

        if (context->Message_Block_Index == 64)
        {
            SHA1ProcessMessageBlock(context, tix);
        }

        message_array++;
    }
}

__device__  void SHA1PadMessage(SHA1Context *context, int tix)
{
    /*
     *  Check to see if the current message block is too small to hold
     *  the initial padding bits and length.  If so, we will pad the
     *  block, process it, and then continue padding into a second
     *  block.
     */
    if (context->Message_Block_Index > 55)
    {
        context->Message_Block[context->Message_Block_Index++] = 0x80;
        while(context->Message_Block_Index < 64)
        {
            context->Message_Block[context->Message_Block_Index++] = 0;
        }

        SHA1ProcessMessageBlock(context, tix);

        while(context->Message_Block_Index < 56)
        {
            context->Message_Block[context->Message_Block_Index++] = 0;
        }
    }
    else
    {
        context->Message_Block[context->Message_Block_Index++] = 0x80;
        while(context->Message_Block_Index < 56)
        {
            context->Message_Block[context->Message_Block_Index++] = 0;
        }
    }

    /*
     *  Store the message length as the last 8 octets
     */
    context->Message_Block[56] = (context->Length_High >> 24) & 0xFF;
    context->Message_Block[57] = (context->Length_High >> 16) & 0xFF;
    context->Message_Block[58] = (context->Length_High >> 8) & 0xFF;
    context->Message_Block[59] = (context->Length_High) & 0xFF;
    context->Message_Block[60] = (context->Length_Low >> 24) & 0xFF;
    context->Message_Block[61] = (context->Length_Low >> 16) & 0xFF;
    context->Message_Block[62] = (context->Length_Low >> 8) & 0xFF;
    context->Message_Block[63] = (context->Length_Low) & 0xFF;

    SHA1ProcessMessageBlock(context, tix);
}

__device__  int SHA1Result(SHA1Context *context, int tix)
{

    if (context->Corrupted)
    {
        return 0;
    }

    if (!context->Computed)
    {
        SHA1PadMessage(context, tix);
        context->Computed = 1;
    }

    return 1;
}


__global__ void hmacsetup(char * cu_shared_key_i, char * cu_shared_key_o){
	//return;
	int j,tix=threadIdx.x;
		tix=blockIdx.x*blockDim.x + threadIdx.x;

	if (tix<=RECEIVERS) {	
		for(j=0;j<64;j++) {
			*(((char *)cu_shared_key_i)+(tix*65+j))^= 0x36;
			*(((char *)cu_shared_key_o)+(tix*65+j))^= 0x5c;
		}
	}
	return;
}


__global__ void hmaccompute(char *cu_message, char * cu_shared_key_i, char * cu_shared_key_o, unsigned *md, int size){


	__shared__ SHA1Context sha[RECEIVERS];       
	int tix=threadIdx.x,j;
	tix=blockIdx.x*blockDim.x + threadIdx.x;
	unsigned temp[5];
/*	__shared__ char message[RECEIVERS][FILESIZE];
	
	int i;

	for(i=0; i<FILESIZE; i++) {
			message[tix][i]=cu_message[i];
	}
	*/
			
		SHA1Reset(&sha[tix]);  
		SHA1Input(&sha[tix], (char *)(cu_shared_key_i)+(tix*65), 64, tix);
		SHA1Input(&sha[tix], cu_message,size, tix);
		SHA1Result(&sha[tix], tix);

		for(j=0; j<5; j++) {
			temp[j]=sha[tix].Message_Digest[j];
		}
		SHA1Reset(&sha[tix]);
		SHA1Input(&sha[tix], (char *)(cu_shared_key_o)+(tix*65), 64, tix);
		SHA1Input(&sha[tix], (char *)temp,20, tix);
		SHA1Result(&sha[tix], tix);	

		for(j=0; j<5; j++) {
			*((unsigned *)(md)+((tix*5)+j))=sha[tix].Message_Digest[j];
		}		
		*((unsigned *)(md)+((tix*5)+4))=tix;
	
	//  __syncthreads();
}


long long hmac_sign_gpu(char *message_files[])
{
    FILE *f_key, *fp;                
    int i, times;
	unsigned char message[FILESIZE+1];
	char *cu_message, *cu_shared_key_i, *cu_shared_key_o;
	unsigned *cu_md,*md;
	char  s_val[RECEIVERS][65];
	
	struct timeval s_time,e_time;
	
	// create the cuda timer
	unsigned int timer=0;
	//CUT_SAFE_CALL(cutCreateTimer(&timer));
	
	md=(unsigned *)malloc(4*RECEIVERS*5);
	
	
	f_key=fopen("shared_key","r");
	
	for (i=0;i<RECEIVERS;i++)
	{
		fscanf(f_key,"%s\n",s_val[i]);
	}
	fclose(f_key);

	if (!(fp = fopen(message_files[1],"r"))) {
		perror("Error in opening file \n");
		exit(0);
	}
	for(i=0;i<FILESIZE;i++)
	{
		message[i]=fgetc(fp);
	}
	fclose(fp);

/*
	printf("\nRunning test on GPU:\n");
	printf("----------------------\n");
	printf("Message Length  : %d bytes\n",FILESIZE);
	printf("No of Messages  : %d \n",NO_OF_MESSAGES);
	printf("No of Receivers : %d \n",RECEIVERS);
	printf("\n");
*/
	hipMalloc( (void **) &cu_shared_key_i, RECEIVERS*65);	
	hipMalloc( (void **) &cu_shared_key_o, RECEIVERS*65);	
	hipMemcpy( (void *) cu_shared_key_i, (void *) s_val, RECEIVERS*65, hipMemcpyHostToDevice );
	hipMemcpy( (void *) cu_shared_key_o, (void *) s_val, RECEIVERS*65, hipMemcpyHostToDevice );

	hmacsetup <<<1,RECEIVERS>>>(cu_shared_key_i, cu_shared_key_o);
	
	hipMalloc( (void **) &cu_message, FILESIZE);
	hipMalloc( (void **) &cu_md, (4*RECEIVERS*5));
	
	hipDeviceSynchronize();
	
	gettimeofday(&s_time,NULL);
	//CUT_SAFE_CALL(cutStartTimer(timer));

	for(times=0; times<NO_OF_MESSAGES; times++) {
	
		CUDA_SAFE_CALL(hipMemcpy( (void *) cu_message, (void *) message, FILESIZE, hipMemcpyHostToDevice ));

		hmaccompute <<<1,RECEIVERS>>>(cu_message, cu_shared_key_i, cu_shared_key_o, cu_md,FILESIZE);
		
		CUDA_SAFE_CALL(hipMemcpy( (void *) md, (void *) cu_md, 4*RECEIVERS*5, hipMemcpyDeviceToHost ));

		hipDeviceSynchronize();

		#ifdef PRINTF
		for(i=0; i<RECEIVERS; i++)
			printf( "%08X %08X %08X %08X %08X\n",md[0+i*5],md[1+i*5],md[ 2 +i *5],md[3 +i*5],md[4+ i *5]);
		printf("\n");
		#endif
		
	}	
	//CUT_SAFE_CALL(cutStopTimer(timer));
	gettimeofday(&e_time,NULL);

	//printf("GPU : Time counted on CPU :  %lld microseconds\n", timeval_diff(NULL,&e_time,&s_time));		
	//printf("GPU : Time counted on GPU :  %.0f microseconds\n",(cutGetTimerValue(timer))*1000);

	hipFree(cu_md);
	hipFree(cu_message);

	return timeval_diff(NULL,&e_time,&s_time);
}


long long hmac_sign_cpu_gpu(char *message_files[])
{
    SHA1Context sha;
    FILE *f_key, *fp;                
    int i, times;
	unsigned char message[FILESIZE+1];
	char *cu_message, *cu_shared_key_i, *cu_shared_key_o;
	unsigned *cu_md,*md;
	char  s_val[RECEIVERS][65];
	
	struct timeval s_time,e_time;
	
	// create the cuda timer
	unsigned int timer=0;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	
	md=(unsigned *)malloc(4*RECEIVERS*5);
	
	
	f_key=fopen("shared_key","r");
	
	for (i=0;i<RECEIVERS;i++)
	{
		fscanf(f_key,"%s\n",s_val[i]);
	}
	fclose(f_key);

	if (!(fp = fopen(message_files[1],"r"))) {
		perror("Error in opening file \n");
		exit(0);
	}
	for(i=0;i<FILESIZE;i++)
	{
		message[i]=fgetc(fp);
	}
	fclose(fp);


	hipMalloc( (void **) &cu_shared_key_i, RECEIVERS*65);	
	hipMalloc( (void **) &cu_shared_key_o, RECEIVERS*65);	
	hipMemcpy( (void *) cu_shared_key_i, (void *) s_val, RECEIVERS*65, hipMemcpyHostToDevice );
	hipMemcpy( (void *) cu_shared_key_o, (void *) s_val, RECEIVERS*65, hipMemcpyHostToDevice );

	hmacsetup <<<1,RECEIVERS>>>(cu_shared_key_i, cu_shared_key_o);

	hipMalloc( (void **) &cu_md, (4*RECEIVERS*5));
	hipDeviceSynchronize();
	hipMalloc( (void **) &cu_message, 20);
	
	gettimeofday(&s_time,NULL);
	CUT_SAFE_CALL(cutStartTimer(timer));

	for(times=0; times<NO_OF_MESSAGES; times++) {

		SHA1Reset_cpu(&sha);
		SHA1Input_cpu(&sha, (char *)message,FILESIZE);
		if (!SHA1Result_cpu(&sha))
		{
			fprintf(stderr,"sha: could not compute message digest for %s\n", message_files[1]);
		}
		else
		{
			for(i=0;i<5;i++) {
				*(((unsigned *)message)+i)=sha.Message_Digest[0];
			}
		}
	
		CUDA_SAFE_CALL(hipMemcpy( (void *) cu_message, (void *) message, 20, hipMemcpyHostToDevice ));

		hmaccompute <<<1,RECEIVERS>>>(cu_message, cu_shared_key_i, cu_shared_key_o, cu_md, 20);
		
		CUDA_SAFE_CALL(hipMemcpy( (void *) md, (void *) cu_md, 4*RECEIVERS*5, hipMemcpyDeviceToHost ));

		hipDeviceSynchronize();

		#ifdef PRINTF
		for(i=0; i<RECEIVERS; i++)
			printf( "%08X %08X %08X %08X %08X\n",md[0+i*5],md[1+i*5],md[ 2 +i *5],md[3 +i*5],md[4+ i *5]);
		printf("\n");
		#endif
		
	}	
	CUT_SAFE_CALL(cutStopTimer(timer));
	gettimeofday(&e_time,NULL);

	//printf("GPU : Time counted on CPU :  %lld microseconds\n", timeval_diff(NULL,&e_time,&s_time));		
	//printf("GPU : Time counted on GPU :  %.0f microseconds\n",(cutGetTimerValue(timer))*1000);

	hipFree(cu_md);
	hipFree(cu_message);

	return timeval_diff(NULL,&e_time,&s_time);
}



__global__ void horse_public_key_compute(char *cu_public, char *cu_private) {

	SHA1Context sha;       
	
	int tix=threadIdx.x,j;
	tix=blockIdx.x*blockDim.x + threadIdx.x;
	
		SHA1Reset(&sha);  
		SHA1Input(&sha, (char *)(cu_private)+(tix*4), 4, tix);
		SHA1Result(&sha, tix);

		for(j=0; j<5; j++) {
			*((unsigned *)(cu_public)+((tix*5)+j))=sha.Message_Digest[j];
		}		
	//	*((unsigned *)(cu_public)+((tix*5)+4))=tix;
	
	//  __syncthreads();*/
}


long long hors_gen_public_key_gpu(){

	int i;
	FILE *f_private, *f_public;
	char buf[20];
	struct timeval s_time,e_time;
	unsigned *private_key;
	unsigned *md;
	char *cu_public, *cu_private;
	
	md=(unsigned *)malloc(4*1024*5);
	private_key=(unsigned *)malloc(4*1024);

	
	f_private=fopen("hors_private_key","r");
	if (f_private == NULL)
	{
		perror("Hors private Key - Error");
		exit(0);
	}

	f_public=fopen("hors_public_key_gpu","w");
	if (f_public == NULL)
	{
		perror("Hors Public Key - Error");
		exit(0);
	}

	gettimeofday(&s_time,NULL);

	hipMalloc( (void **) &cu_public, (4*1024*5));
	hipMalloc( (void **) &cu_private, (4*1024));


	fscanf(f_private,"%s\n",buf);
	fprintf(f_public,"%s\n",buf);
	for (i=0;i<1024;i++)
	{
		fscanf(f_private,"%s\n",buf);
		private_key[i]=atol(buf);
		//printf("%ld\n", private_key[i]);
	}
	
	CUDA_SAFE_CALL(hipMemcpy( (void *) cu_private, (void *) private_key, 1024 * sizeof(unsigned), hipMemcpyHostToDevice ));

	horse_public_key_compute <<<2,512>>>(cu_public, cu_private);

	CUDA_SAFE_CALL(hipMemcpy( (void *) md, (void *) cu_public, 4*1024*5, hipMemcpyDeviceToHost));

	for(i=0; i<1024; i++)
		fprintf( f_public, "%08X%08X%08X%08X%08X\n",md[0+i*5],md[1+i*5],md[ 2 +i *5],md[3 +i*5],md[4+ i *5]);

	gettimeofday(&e_time,NULL);

	fclose(f_private);
	fclose(f_public);
	return timeval_diff(NULL,&e_time,&s_time);
}


__global__ void horse_sign(char *cu_md, char * cu_h_val) {


	int tix=threadIdx.x,j;
	tix=blockIdx.x*blockDim.x + threadIdx.x;

	switch(tix){
		case 0:
			*((int *)(cu_h_val)+tix)=(*(unsigned *)(cu_md)+0 )>>22 & 0x000003FF;
			break;
		case 1:
			*((int *)(cu_h_val)+tix)=(*(unsigned *)(cu_md)+0 )>>12 & 0x000003FF;
			break;
		case 2:
			*((int *)(cu_h_val)+tix)=(*(unsigned *)(cu_md)+0 )>>2 & 0x000003FF;
			break;
		case 3:
			*((int *)(cu_h_val)+tix)=(((*(unsigned *)(cu_md)+0 ) & 0x00000003)<<8) + ((*((unsigned *)(cu_md)+1) )>>24 & 0x000000FF);
			break;
		case 4:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+1 )>>14 & 0x000003FF;
			break;
		case 5:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+1 )>>4 & 0x000003FF;
			break;
		case 6:
			*((int *)(cu_h_val)+tix)=((*((unsigned *)(cu_md)+1)  & 0x0000000F)<<6 )+ (*((unsigned *)(cu_md)+2 )>>26 & 0x0000003F);
			break;
		case 7:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+2 )>>16 & 0x000003FF;
			break;
		case 8:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+2 )>>6 & 0x000003FF;
			break;
		case 9:
			*((int *)(cu_h_val)+tix)=((*((unsigned *)(cu_md)+2)  & 0x0000000F)<<4 )+ (*((unsigned *)(cu_md)+3 )>>28 & 0x0000000F);
			break;
		case 10:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+3 )>>18 & 0x000003FF;
			break;		
		case 11:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+3 )>>8 & 0x000003FF;
			break;
		case 12:
			*((int *)(cu_h_val)+tix)=((*((unsigned *)(cu_md)+3)  & 0x000000FF)<<2 )+ (*((unsigned *)(cu_md)+4 )>>30 & 0x00000003);
			break;
		case 13:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+4 )>>20 & 0x000003FF;
			break;		
		case 14:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+4 )>>10 & 0x000003FF;
			break;
		case 15:
			*((int *)(cu_h_val)+tix)=*((unsigned *)(cu_md)+4 )>>0 & 0x000003FF;
			break;
		default:
			break;
	}
}


long long hors_sign_gpu(char *message_files[]) {

    SHA1Context sha;
    int i, times;
	FILE *fp,*f_key,*f_sign;
	char message[FILESIZE];
	int h_val[16];
	char *cu_md, *cu_h_val;
	struct timeval s_time,e_time;

	if (!(fp = fopen(message_files[1],"r"))) {
		perror("Error in opening file \n");
		exit(0);
	}

	for(i=0;i<FILESIZE;i++)
	{
		message[i]=fgetc(fp);
	}
	fclose(fp);

	char  s_val[1024][11];
	f_key=fopen("hors_private_key","r");
	
	fscanf(f_key,"%s\n",s_val[0]);
	for (i=0;i<1024;i++)
	{
		fscanf(f_key,"%s\n",s_val[i]);
	}


	gettimeofday(&s_time,NULL);

	for(times=0; times<NO_OF_MESSAGES; times++) {
		SHA1Reset_cpu(&sha);

		SHA1Input_cpu(&sha, (char *)message,FILESIZE);
		
		if (!SHA1Result_cpu(&sha))
		{
			fprintf(stderr, "ERROR-- could not compute message digest\n");
			exit(0);
		}

		#ifdef PRINTF
		int j;
		printf("SHA1 of '%s' = ",message_files[1]);
		for(j = 0; j < 5 ; j++)
		{
			printf("%08X ",sha.Message_Digest[j]);
		}
		printf("\n");
		#endif

		hipMalloc( (void **) &cu_md, (4*5));
		hipMalloc( (void **) &cu_h_val, (16*sizeof(int)));
		CUDA_SAFE_CALL(hipMemcpy( (void *) cu_md, (void *) sha.Message_Digest, 5 * sizeof(unsigned), hipMemcpyHostToDevice ));

		horse_sign<<<1,16>>>(cu_md,cu_h_val);

		CUDA_SAFE_CALL(hipMemcpy( (void *) h_val, (void *) cu_h_val, (16*sizeof(int)), hipMemcpyDeviceToHost));

		fclose(f_key);
		
	//	f_sign=fopen("hors_signature_gpu","w");
	#ifdef PRINTF
		for(i=0;i<16;i++) {
			printf("%s\n",s_val[h_val[i]-1]);
		//	fprintf(f_sign,"%s\n",s_val[h_val[i]-1]);
		}
	//	fclose(f_sign);
	#endif
	}
	gettimeofday(&e_time,NULL);

	return timeval_diff(NULL,&e_time,&s_time);
}


__global__ void horse_verify(int * cu_result, char * cu_public_val, char * cu_sign, unsigned *cu_md) {

	__shared__ SHA1Context sha[16];

	int tix=threadIdx.x,j;
	tix=blockIdx.x*blockDim.x + threadIdx.x;
	*cu_result=1;
	__syncthreads();
	
	SHA1Reset(&sha[tix]);  
	SHA1Input(&sha[tix], (char *)((unsigned *)(cu_sign)+(tix)), 4, tix);
	SHA1Result(&sha[tix], tix);

//	for(j=0; j<5; j++) {
//			*((unsigned *)(cu_md)+((tix*5)+j))=sha[tix].Message_Digest[j];
//	}	
//	*((unsigned *)(cu_md)+((tix*5)+4))=tix;
//	*((unsigned *)(cu_md)+((tix*5)+3))=*(((unsigned *)cu_sign) + tix);
	if(*((unsigned *)(cu_public_val+(tix*5*4))+0)!=sha[tix].Message_Digest[0]
			|| *((unsigned *)(cu_public_val+(tix*5*4))+1)!=sha[tix].Message_Digest[1]
			|| *((unsigned *)(cu_public_val+(tix*5*4))+2)!=sha[tix].Message_Digest[2]
			|| *((unsigned *)(cu_public_val+(tix*5*4))+3)!=sha[tix].Message_Digest[3]
			|| *((unsigned *)(cu_public_val+(tix*5*4))+4)!=sha[tix].Message_Digest[4]) {
				
		*cu_result=-1;
	}
}


long long hors_verify_gpu(char *message_files[]) {

	std::stringstream ss;
    SHA1Context sha;
    int i,j, result=0, times;
	FILE *fp,*f_key,*f_sign;
	char message[FILESIZE];
	int h_val[16];
	struct timeval s_time,e_time;
	char  s_val[1024][41];
	unsigned public_val[16][5];
	char buf[41];
	unsigned signature[16];
	char *cu_sign, *cu_public_val;
	int *cu_result;
	
	unsigned *cu_md,*md;
	md=(unsigned *)malloc(4*16*5);

	if (!(fp = fopen(message_files[1],"r"))) {
		perror("Error in opening file \n");
		exit(0);
	}
	for(i=0;i<FILESIZE;i++)
	{
		message[i]=fgetc(fp);
	}
	fclose(fp);


	f_key=fopen("hors_public_key_gpu","r");
	fscanf(f_key,"%s\n",s_val[0]);
	for (i=0;i<1024;i++)
	{
		fscanf(f_key,"%s\n",s_val[i]);
	}
	fclose(f_key);


	f_sign=fopen("hors_signature_gpu","r");
	for (i=0;i<16;i++)
	{
		fscanf(f_sign,"%s\n",buf);
		signature[i]=atol(buf);
	}
	fclose(f_sign);

	gettimeofday(&s_time,NULL);
	
	for(times=0; times<NO_OF_MESSAGES; times++) {
		
		SHA1Reset_cpu(&sha);
		SHA1Input_cpu(&sha, (char *)message,FILESIZE);

		if (!SHA1Result_cpu(&sha))
		{
			fprintf(stderr, "ERROR-- could not compute message digest\n");
			exit(0);
		}
		
		#ifdef PRINTF
		printf("SHA1 of '%s' = ",message_files[1]);
		for(j = 0; j < 5 ; j++)
		{
				printf("%08X ",sha.Message_Digest[j]);
		}
		printf("\n");
		#endif

		h_val[0] = (sha.Message_Digest[0]>>22) & 0x000003FF;
		h_val[1] = (sha.Message_Digest[0]>>12) & 0x000003FF;
		h_val[2] = (sha.Message_Digest[0]>>2) & 0x000003FF;
		h_val[3] = (((sha.Message_Digest[0]) & 0x00000003) << 8)+ ((sha.Message_Digest[1]>>24) & 0x000000FF );
		h_val[4] = (sha.Message_Digest[1]>>14) & 0x000003FF;
		h_val[5] = (sha.Message_Digest[1]>>4) & 0x000003FF;
		h_val[6] = (((sha.Message_Digest[1]) & 0x0000000F) << 6)+ ((sha.Message_Digest[2]>>26) & 0x0000003F );
		h_val[7] = (sha.Message_Digest[2]>>16) & 0x000003FF;
		h_val[8] = (sha.Message_Digest[2]>>6) & 0x000003FF;
		h_val[9] = (((sha.Message_Digest[2]) & 0x0000003F) << 4)+ ((sha.Message_Digest[3]>>28) & 0x0000000F );
		h_val[10]= (sha.Message_Digest[3]>>18) & 0x000003FF;
		h_val[11]= (sha.Message_Digest[3]>>8) & 0x000003FF;
		h_val[12]= (((sha.Message_Digest[3]) & 0x000000FF) << 2)+ ((sha.Message_Digest[4]>>30) & 0x00000003 );
		h_val[13]= (sha.Message_Digest[4]>>20) & 0x000003FF;
		h_val[14]= (sha.Message_Digest[4]>>10) & 0x000003FF;
		h_val[15]= (sha.Message_Digest[4]) & 0x000003FF;

		for(i=0; i<16; i++) {
			#ifdef PRINTF
			puts(s_val[h_val[i]-1]);
			#endif
			strcpy(buf,s_val[h_val[i]-1]);
			public_val[i][4] = strtoul( buf+32, NULL, 16 ); 
			buf[32]='\0';
			public_val[i][3] = strtoul( buf+24, NULL, 16 ); 
			buf[24]='\0';
			public_val[i][2] = strtoul( buf+16, NULL, 16 ); 
			buf[16]='\0';
			public_val[i][1] = strtoul( buf+8, NULL, 16 ); 
			buf[8]='\0';
			public_val[i][0] = strtoul( buf, NULL, 16 ); 
		}
		
		hipMalloc( (void **) &cu_md, (4*16*5));

		
		hipMalloc( (void **) &cu_result, sizeof(int));
		hipMalloc( (void **) &cu_public_val, 16 * 5 * sizeof(unsigned));
		hipMalloc( (void **) &cu_sign, sizeof(unsigned)*16);
		CUDA_SAFE_CALL(hipMemcpy( (void *) cu_sign, (void *) signature, 16 * sizeof(unsigned), hipMemcpyHostToDevice ));
		CUDA_SAFE_CALL(hipMemcpy( (void *) cu_public_val, (void *) public_val, 16 * 5 * sizeof(unsigned), hipMemcpyHostToDevice ));

		horse_verify<<<1,16>>>(cu_result, cu_public_val, cu_sign,cu_md);

		CUDA_SAFE_CALL(hipMemcpy( (void *) &result, (void *) cu_result, sizeof(int), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy( (void *) md, (void *) cu_md, 4*16*5, hipMemcpyDeviceToHost ));

		#ifdef PRINTF
		printf("\n");
		for(i=0; i<16; i++)
			printf( "%08X %08X %08X %0ld %08X\n",md[0+i*5],md[1+i*5],md[ 2 +i *5],md[3 +i*5],md[4+ i *5]);
		printf("\n");
		#endif
		
		#ifdef PRINTF
		printf("Result %d\n",result);
		#endif
		if(result!=-1)
			printf("Success - ");
		else
			printf("Failed  - ");
	}
	gettimeofday(&e_time,NULL);

	return timeval_diff(NULL,&e_time,&s_time);

}





